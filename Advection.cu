#include "hip/hip_runtime.h"
//#include "wr.h"
#include <hipcub/hipcub.hpp>
#include <hip/hip_math_constants.h>
#include <math_functions.h>
#include <stdio.h>
#include <cfloat>

#define USE_GPUMANAGER 0
#define USE_SHARED_MEM 0
#define SUB_BLOCK_SIZE 8
#define NUM_DIMS 3

#define gpuSafe(retval) gpuPrintErr((retval), __FILE__, __LINE__)
#define gpuCheck() gpuPrintErr(hipGetLastError(), __FILE__, __LINE__)

inline void gpuPrintErr(hipError_t err, const char *file, int line) {
  if (err != hipSuccess)
    fprintf(stderr,"CUDA Error: %s at %s:%d\n", hipGetErrorString(err), file, line);
}

__global__ void decisionKernel1(float *u, float *delu, float *delua, float dx, float dy, float dz, int block_size) {
#define INDEX(i,j,k) (((k) * (block_size+2) + (j)) * (block_size+2) + (i))
#define INDEX4(d,i,j,k) ((((d) * (block_size+2) + (k)) * (block_size+2) + (j)) * (block_size+2) + (i))
  float delx = 0.5/dx;
  float dely = 0.5/dy;
  float delz = 0.5/dz;
#if USE_SHARED_MEM
  __shared__ float u_s[SUB_BLOCK_SIZE][SUB_BLOCK_SIZE][SUB_BLOCK_SIZE];

  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
#endif
  int gx = blockDim.x * blockIdx.x + threadIdx.x;
  int gy = blockDim.y * blockIdx.y + threadIdx.y;
  int gz = blockDim.z * blockIdx.z + threadIdx.z;

#if USE_SHARED_MEM
  // read u into shared memory
  if ((gx < (block_size + 2)) && (gy < (block_size + 2)) && (gz < (block_size + 2))) {
    u_s[tx][ty][tz] = u[INDEX(gx,gy,gz)];
  }
  __syncthreads();
#endif

  // calculate differentials
  float u_pos, u_neg;
  if (((gx >= 1 && gx <= block_size) && (gy >= 1 && gy <= block_size)) && (gz >= 1 && gz <= block_size)) {
    // d/dx
#if USE_SHARED_MEM
    u_pos = (tx < SUB_BLOCK_SIZE-1) ? (u_s[tx+1][ty][tz]) : (u[INDEX(gx+1,gy,gz)]);
    u_neg = (tx > 0) ? (u_s[tx-1][ty][tz]) : (u[INDEX(gx-1,gy,gz)]);
#else
    u_pos = u[INDEX(gx+1,gy,gz)];
    u_neg = u[INDEX(gx-1,gy,gz)];
#endif
    delu[INDEX4(0,gx,gy,gz)] = (u_pos - u_neg)*delx;
    delua[INDEX4(0,gx,gy,gz)] = (fabsf(u_pos) + fabsf(u_neg))*delx;

    // d/dy
#if USE_SHARED_MEM
    u_pos = (ty < SUB_BLOCK_SIZE-1) ? (u_s[tx][ty+1][tz]) : (u[INDEX(gx,gy+1,gz)]);
    u_neg = (ty > 0) ? (u_s[tx][ty-1][tz]) : (u[INDEX(gx,gy-1,gz)]);
#else
    u_pos = u[INDEX(gx,gy+1,gz)];
    u_neg = u[INDEX(gx,gy-1,gz)];
#endif
    delu[INDEX4(1,gx,gy,gz)] = (u_pos - u_neg)*dely;
    delua[INDEX4(1,gx,gy,gz)] = (fabsf(u_pos) + fabsf(u_neg))*dely;

    // d/dz
#if USE_SHARED_MEM
    u_pos = (tz < SUB_BLOCK_SIZE-1) ? (u_s[tx][ty][tz+1]) : (u[INDEX(gx,gy,gz+1)]);
    u_neg = (tz > 0) ? (u_s[tx][ty][tz-1]) : (u[INDEX(gx,gy,gz-1)]);
#else
    u_pos = u[INDEX(gx,gy,gz+1)];
    u_neg = u[INDEX(gx,gy,gz-1)];
#endif
    delu[INDEX4(2,gx,gy,gz)] = (u_pos - u_neg)*delz;
    delua[INDEX4(2,gx,gy,gz)] = (fabsf(u_pos) + fabsf(u_neg))*delz;
  }
#undef INDEX
#undef INDEX4
}

__global__ void decisionKernel2(float *delu, float *delua, float *delu_n_g, float *errors, float refine_filter, float dx, float dy, float dz, int block_size) {
#define INDEX4(d,i,j,k) ((((d) * (block_size+2) + (k)) * (block_size+2) + (j)) * (block_size+2) + (i))
#define INDEX4C(i,j,k,d) ((((d) * (block_size-2) + (k)) * (block_size-2) + (j)) * (block_size-2) + (i))
#define ERR_INDEX(i,j,k) ((((k)-2) * (block_size-2) + ((j)-2)) * (block_size-2) + ((i)-2))
  float delx = 0.5/dx;
  float dely = 0.5/dy;
  float delz = 0.5/dz;
  float delu_n[3][NUM_DIMS * NUM_DIMS];
#if USE_SHARED_MEM
  __shared__ float delu_s[NUM_DIMS][SUB_BLOCK_SIZE][SUB_BLOCK_SIZE][SUB_BLOCK_SIZE];
  __shared__ float delua_s[NUM_DIMS][SUB_BLOCK_SIZE][SUB_BLOCK_SIZE][SUB_BLOCK_SIZE];
#endif

#if 0 //USE_SHARED_MEM
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
#endif
  int gx = blockDim.x * blockIdx.x + threadIdx.x + 1;
  int gy = blockDim.y * blockIdx.y + threadIdx.y + 1;
  int gz = blockDim.z * blockIdx.z + threadIdx.z + 1;

#if 0 //USE_SHARED_MEM
  // read delu & delua into shared memory
  if (gx <= block_size && gy <= block_size && gz <= block_size) {
    for (int d = 0; d < NUM_DIMS; d++) {
      delu_s[d][tx][ty][tz] = delu[INDEX4(d,gx,gy,gz)];
      delua_s[d][tx][ty][tz] = delua[INDEX4(d,gx,gy,gz)];
    }
  }
  __syncthreads();
#endif

  // calculate error per thread
  float delu_pos, delu_neg;
  float delua_pos, delua_neg;
  float num = 0., denom = 0.;
  float error;
  if ((gx > 1 && gx < block_size) && (gy > 1 && gy < block_size) && (gz > 1 && gz < block_size)) {
    for (int d = 0; d < NUM_DIMS; d++) {
#if 0 //USE_SHARED_MEM
      delu_pos = (tx < SUB_BLOCK_SIZE-1) ? (delu_s[d][tx+1][ty][tz]) : (delu[INDEX4(d,gx+2,gy+1,gz+1)]);
      delu_neg = (tx > 0) ? (delu_s[d][tx-1][ty][tz]) : (delu[INDEX4(d,gx,gy+1,gz+1)]);
      delua_pos = (tx < SUB_BLOCK_SIZE-1) ? (delua_s[d][tx+1][ty][tz]) : (delua[INDEX4(d,gx+2,gy+1,gz+1)]);
      delua_neg = (tx > 0) ? (delua_s[d][tx-1][ty][tz]) : (delua[INDEX4(d,gx,gy+1,gz+1)]);
#else
      delu_pos = delu[INDEX4(d,gx+1,gy,gz)];
      delu_neg = delu[INDEX4(d,gx-1,gy,gz)];
      delua_pos = delua[INDEX4(d,gx+1,gy,gz)];
      delua_neg = delua[INDEX4(d,gx-1,gy,gz)];
#endif
      delu_n[0][3*d+0] = (delu_pos - delu_neg)*delx;
      delu_n[1][3*d+0] = (fabsf(delu_pos) + fabsf(delu_neg))*delx;
      delu_n[2][3*d+0] = (delua_pos + delua_neg)*delx;
      delu_n_g[INDEX4C(gx-2,gy-2,gz-2,3*d+0)] = delu_n[0][3*d+0];
      delu_n_g[INDEX4C(gx-2,gy-2,gz-2,9+3*d+0)] = delu_n[1][3*d+0];
      delu_n_g[INDEX4C(gx-2,gy-2,gz-2,18+3*d+0)] = delu_n[2][3*d+0];
      /*
      if (gx == 12 && gy == 12 && gz == 9 && (9+3*d+0) == 9) {
        printf("[GPU] delu_pos: %f, delu_neg: %f\n", delu_pos, delu_neg);
        printf("[GPU] delua_pos: %f, delua_neg: %f\n", delua_pos, delua_neg);
        printf("[GPU] delu_n: %f\n", delu_n[1][3*d+0]);
      }
      */

#if 0 //USE_SHARED_MEM
      delu_pos = (ty < SUB_BLOCK_SIZE-1) ? (delu_s[d][tx][ty+1][tz]) : (delu[INDEX4(d,gx+1,gy+2,gz+1)]);
      delu_neg = (ty > 0) ? (delu_s[d][tx][ty-1][tz]) : (delu[INDEX4(d,gx+1,gy,gz+1)]);
      delua_pos = (ty < SUB_BLOCK_SIZE-1) ? (delua_s[d][tx][ty+1][tz]) : (delua[INDEX4(d,gx+1,gy+2,gz+1)]);
      delua_neg = (ty > 0) ? (delua_s[d][tx][ty-1][tz]) : (delua[INDEX4(d,gx+1,gy,gz+1)]);
#else
      delu_pos = delu[INDEX4(d,gx,gy+1,gz)];
      delu_neg = delu[INDEX4(d,gx,gy-1,gz)];
      delua_pos = delua[INDEX4(d,gx,gy+1,gz)];
      delua_neg = delua[INDEX4(d,gx,gy-1,gz)];
#endif
      delu_n[0][3*d+1] = (delu_pos - delu_neg)*dely;
      delu_n[1][3*d+1] = (fabsf(delu_pos) + fabsf(delu_neg))*dely;
      delu_n[2][3*d+1] = (delua_pos + delua_neg)*dely;
      delu_n_g[INDEX4C(gx-2,gy-2,gz-2,3*d+1)] = delu_n[0][3*d+1];
      delu_n_g[INDEX4C(gx-2,gy-2,gz-2,9+3*d+1)] = delu_n[1][3*d+1];
      delu_n_g[INDEX4C(gx-2,gy-2,gz-2,18+3*d+1)] = delu_n[2][3*d+1];

#if 0 //USE_SHARED_MEM
      delu_pos = (tz < SUB_BLOCK_SIZE-1) ? (delu_s[d][tx][ty][tz+1]) : (delu[INDEX4(d,gx+1,gy+1,gz+2)]);
      delu_neg = (tz > 0) ? (delu_s[d][tx][ty][tz-1]) : (delu[INDEX4(d,gx+1,gy+1,gz)]);
      delua_pos = (tz < SUB_BLOCK_SIZE-1) ? (delua_s[d][tx][ty][tz+1]) : (delua[INDEX4(d,gx+1,gy+1,gz+2)]);
      delua_neg = (tz > 0) ? (delua_s[d][tx][ty][tz-1]) : (delua[INDEX4(d,gx+1,gy+1,gz)]);
#else
      delu_pos = delu[INDEX4(d,gx,gy,gz+1)];
      delu_neg = delu[INDEX4(d,gx,gy,gz-1)];
      delua_pos = delua[INDEX4(d,gx,gy,gz+1)];
      delua_neg = delua[INDEX4(d,gx,gy,gz-1)];
#endif
      delu_n[0][3*d+2] = (delu_pos - delu_neg)*delz;
      delu_n[1][3*d+2] = (fabsf(delu_pos) + fabsf(delu_neg))*delz;
      delu_n[2][3*d+2] = (delua_pos + delua_neg)*delz;
      delu_n_g[INDEX4C(gx-2,gy-2,gz-2,3*d+2)] = delu_n[0][3*d+2];
      delu_n_g[INDEX4C(gx-2,gy-2,gz-2,9+3*d+2)] = delu_n[1][3*d+2];
      delu_n_g[INDEX4C(gx-2,gy-2,gz-2,18+3*d+2)] = delu_n[2][3*d+2];
    }

    for (int dd = 0; dd < NUM_DIMS * NUM_DIMS; dd++) {
      num = num + powf(delu_n[0][dd], 2.);
      denom = denom + powf(delu_n[1][dd], 2.) + (refine_filter * delu_n[2][dd]) * 2;
    }

    if (denom == 0. && num != 0.) {
      printf("D denom is zero!!!!!!!!!!!!!!!!!!!\n");
      error = FLT_MAX;
    }
    else if (denom != 0.0) {
      error = fmaxf(error, num/denom);
    }
    //if (error > 1)
    //  printf("error: %f\n", error);

    // store error in global memory
    errors[ERR_INDEX(gx,gy,gz)] = error;
  }
#undef INDEX4
#undef INDEX4C
#undef ERR_INDEX
}

float invokeDecisionKernel(float *u, float *delu_n, float refine_filter, float dx, float dy, float dz, int block_size) {
  float error;
#if !USE_GPUMANAGER
  float *h_error;
  gpuSafe(hipHostMalloc(&h_error, sizeof(float)));
  float *h_delu_n;
  size_t delu_n_size = sizeof(float)*(block_size-2)*(block_size-2)*(block_size-2)*3*NUM_DIMS*NUM_DIMS;
  gpuSafe(hipHostMalloc(&h_delu_n, delu_n_size));

  hipStream_t decisionStream;
  float *d_error, *d_errors;
  float *d_u, *d_delu, *d_delua;
  size_t u_size = sizeof(float)*(block_size+2)*(block_size+2)*(block_size+2);
  size_t delu_size = NUM_DIMS * u_size;
  size_t errors_size = sizeof(float)*(block_size-2)*(block_size-2)*(block_size-2);
  float *d_delu_n;

  gpuSafe(hipStreamCreate(&decisionStream));
  gpuSafe(hipMalloc(&d_u, u_size));
  gpuSafe(hipMalloc(&d_delu, delu_size));
  gpuSafe(hipMalloc(&d_delua, delu_size));
  gpuSafe(hipMalloc(&d_error, sizeof(float)));
  gpuSafe(hipMemset(d_error, 0, sizeof(float)));
  gpuSafe(hipMalloc(&d_errors, errors_size));
  gpuSafe(hipMalloc(&d_delu_n, delu_n_size));

  gpuSafe(hipMemcpyAsync(d_u, u, u_size, hipMemcpyHostToDevice, decisionStream));

  int sub_block_cnt = ceil((float)(block_size+2)/SUB_BLOCK_SIZE);
  dim3 dimGrid(sub_block_cnt, sub_block_cnt, sub_block_cnt);
  dim3 dimBlock(SUB_BLOCK_SIZE, SUB_BLOCK_SIZE, SUB_BLOCK_SIZE);
  decisionKernel1<<<dimGrid, dimBlock, 0, decisionStream>>>(d_u, d_delu, d_delua, dx, dy, dz, block_size);
  gpuCheck();

  sub_block_cnt = ceil((float)block_size/SUB_BLOCK_SIZE);
  dimGrid = dim3(sub_block_cnt, sub_block_cnt, sub_block_cnt);
  decisionKernel2<<<dimGrid, dimBlock, 0, decisionStream>>>(d_delu, d_delua, d_delu_n, d_errors, refine_filter, dx, dy, dz, block_size);
  gpuCheck();

  gpuSafe(hipMemcpyAsync(h_delu_n, d_delu_n, delu_n_size, hipMemcpyDeviceToHost, decisionStream));

  gpuSafe(hipDeviceSynchronize());
  memcpy(delu_n, h_delu_n, delu_n_size);

  // max reduction using cub
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_errors, d_error, (block_size-2)*(block_size-2)*(block_size-2));
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_errors, d_error, (block_size-2)*(block_size-2)*(block_size-2));

  gpuSafe(hipMemcpyAsync(h_error, d_error, sizeof(float), hipMemcpyDeviceToHost, decisionStream));

  gpuSafe(hipDeviceSynchronize());

  error = *h_error;

  gpuSafe(hipFree(d_u));
  gpuSafe(hipFree(d_delu));
  gpuSafe(hipFree(d_delua));
  gpuSafe(hipFree(d_error));
  gpuSafe(hipFree(d_errors));
  gpuSafe(hipFree(d_delu_n));
  gpuSafe(hipFree(d_temp_storage));
  gpuSafe(hipHostFree(h_error));
  gpuSafe(hipHostFree(h_delu_n));

  gpuSafe(hipStreamDestroy(decisionStream));
#else

#endif

  return error;
}
